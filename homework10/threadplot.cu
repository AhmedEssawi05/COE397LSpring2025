#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//------------------------------------------------------------------------------
// Kernel: each thread writes its global linear index (or ID) into d_array.
//
__global__ void fillGlobalIndex2D(int *d_array, int totalX, int totalY)
{
    // Compute each thread's 2D coordinates in the overall grid
    int globalX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalY = blockIdx.y * blockDim.y + threadIdx.y;

    // Convert (globalY, globalX) into a single linear index
    // in row-major order:
    //    index = row * width + col
    int idx = globalY * totalX + globalX;

    // Make sure we do not go out of bounds
    if (globalX < totalX && globalY < totalY)
    {
        d_array[idx] = idx;
    }
}

//------------------------------------------------------------------------------
int main(int argc, char *argv[])
{
    // Expect 4 command line arguments: blockDimX blockDimY gridDimX gridDimY
    if (argc < 5) {
        fprintf(stderr, "Usage: %s blockDimX blockDimY gridDimX gridDimY\n", argv[0]);
        return 1;
    }

    // Parse command line arguments
    int blockDimX = atoi(argv[1]);
    int blockDimY = atoi(argv[2]);
    int gridDimX  = atoi(argv[3]);
    int gridDimY  = atoi(argv[4]);

    // Compute total array dimensions
    // (the total number of threads in each dimension)
    int totalX = blockDimX * gridDimX;
    int totalY = blockDimY * gridDimY;
    int totalSize = totalX * totalY;

    printf("blockDim = (%d, %d), gridDim = (%d, %d)\n", 
           blockDimX, blockDimY, gridDimX, gridDimY);
    printf("=> totalX = %d, totalY = %d\n", totalX, totalY);

    // Allocate host memory
    int *h_array = (int*)malloc(totalSize * sizeof(int));
    if (!h_array) {
        fprintf(stderr, "Error: host memory allocation failed.\n");
        return 1;
    }

    // Allocate device memory
    int *d_array = NULL;
    hipError_t err = hipMalloc((void**)&d_array, totalSize * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error: device memory allocation failed: %s\n",
                hipGetErrorString(err));
        free(h_array);
        return 1;
    }

    // Zero out the device memory (optional, but good practice)
    hipMemset(d_array, 0, totalSize * sizeof(int));

    // Prepare 2D execution configuration
    dim3 block(blockDimX, blockDimY);
    dim3 grid(gridDimX, gridDimY);

    // Launch the kernel
    fillGlobalIndex2D<<<grid, block>>>(d_array, totalX, totalY);

    // Check for any kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(err));
        hipFree(d_array);
        free(h_array);
        return 1;
    }

    // Copy results back to host
    err = hipMemcpy(h_array, d_array, totalSize * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy to host failed: %s\n",
                hipGetErrorString(err));
        hipFree(d_array);
        free(h_array);
        return 1;
    }

    // Print the 2D layout of global thread indices
    // h_array[row * totalX + col] should contain the linear index
    printf("\nGlobal thread indices in a 2D layout:\n");
    for (int row = 0; row < totalY; ++row)
    {
        for (int col = 0; col < totalX; ++col)
        {
            int idx = row * totalX + col;
            printf("%4d ", h_array[idx]);
        }
        printf("\n");
    }

    // Clean up
    hipFree(d_array);
    free(h_array);

    return 0;
}
