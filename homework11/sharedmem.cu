#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16  // Adjust as needed

// Kernel for 2D vector (matrix) addition using shared memory
__global__
void vectorAdd2D(const float *A, const float *B, float *C, int numRows, int numCols)
{
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    // Load data from global memory into shared memory if within bounds
    if (row < numRows && col < numCols) {
        tileA[threadIdx.y][threadIdx.x] = A[row * numCols + col];
        tileB[threadIdx.y][threadIdx.x] = B[row * numCols + col];
    } else {
        tileA[threadIdx.y][threadIdx.x] = 0.0f;
        tileB[threadIdx.y][threadIdx.x] = 0.0f;
    }

    __syncthreads();

    // Perform the addition and write the result back to global memory
    if (row < numRows && col < numCols) {
        C[row * numCols + col] = tileA[threadIdx.y][threadIdx.x] + tileB[threadIdx.y][threadIdx.x];
    }
}

int main(void)
{
    // Matrix dimensions (example values)
    int numRows = 1024;
    int numCols = 1024;
    size_t size = numRows * numCols * sizeof(float);

    // Allocate host memory for matrices A, B, and C
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    if (!h_A || !h_B || !h_C) {
        fprintf(stderr, "Host memory allocation failed\n");
        exit(EXIT_FAILURE);
    }

    // Initialize matrices with random values
    for (int i = 0; i < numRows * numCols; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid((numCols + TILE_WIDTH - 1) / TILE_WIDTH,
                 (numRows + TILE_WIDTH - 1) / TILE_WIDTH,
                 1);

    // Set up CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);

    // Launch the kernel
    vectorAdd2D<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, numRows, numCols);

    // Record the stop event and synchronize
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time in milliseconds
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Kernel execution time: %f ms\n", elapsedTime);

    // Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy the result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Optionally verify results here (omitted for brevity)

    // Clean up device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    printf("2D vector addition completed using CUDA shared memory.\n");
    return 0;
}
